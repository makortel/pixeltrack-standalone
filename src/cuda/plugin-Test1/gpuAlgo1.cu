#include "hip/hip_runtime.h"
#include "gpuAlgo1.h"

#include "CUDACore/device_unique_ptr.h"
#include "CUDACore/host_unique_ptr.h"

namespace {
  constexpr int NUM_VALUES = 4000;

  template <typename T>
  __global__ void vectorAdd(const T *a, const T *b, T *c, int numElements) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numElements) {
      c[i] = a[i] + b[i];
    }
  }

  template <typename T>
  __global__ void vectorProd(const T *a, const T *b, T *c, int numElements) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < numElements && col < numElements) {
      c[row * numElements + col] = a[row] * b[col];
    }
  }

  template <typename T>
  __global__ void matrixMul(const T *a, const T *b, T *c, int numElements) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < numElements && col < numElements) {
      T tmp = 0;
      for (int i = 0; i < numElements; ++i) {
        tmp += a[row * numElements + i] * b[i * numElements + col];
      }
      c[row * numElements + col] = tmp;
    }
  }

  template <typename T>
  __global__ void matrixMulVector(const T *a, const T *b, T *c, int numElements) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < numElements) {
      T tmp = 0;
      for (int i = 0; i < numElements; ++i) {
        tmp += a[row * numElements + i] * b[i];
      }
      c[row] = tmp;
    }
  }
}  // namespace

cms::cuda::device::unique_ptr<float[]> gpuAlgo1(hipStream_t stream) {
  auto h_a = cms::cuda::make_host_unique<float[]>(NUM_VALUES, stream);
  auto h_b = cms::cuda::make_host_unique<float[]>(NUM_VALUES, stream);

  for (auto i = 0; i < NUM_VALUES; i++) {
    h_a[i] = i;
    h_b[i] = i * i;
  }

  auto d_a = cms::cuda::make_device_unique<float[]>(NUM_VALUES, stream);
  auto d_b = cms::cuda::make_device_unique<float[]>(NUM_VALUES, stream);

  cudaCheck(hipMemcpyAsync(d_a.get(), h_a.get(), NUM_VALUES * sizeof(float), hipMemcpyHostToDevice, stream));
  cudaCheck(hipMemcpyAsync(d_b.get(), h_b.get(), NUM_VALUES * sizeof(float), hipMemcpyHostToDevice, stream));

  int threadsPerBlock{32};
  int blocksPerGrid = (NUM_VALUES + threadsPerBlock - 1) / threadsPerBlock;

  auto d_c = cms::cuda::make_device_unique<float[]>(NUM_VALUES, stream);
  auto current_device = cms::cuda::currentDevice();
  vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_a.get(), d_b.get(), d_c.get(), NUM_VALUES);

  auto d_ma = cms::cuda::make_device_unique<float[]>(NUM_VALUES * NUM_VALUES, stream);
  auto d_mb = cms::cuda::make_device_unique<float[]>(NUM_VALUES * NUM_VALUES, stream);
  auto d_mc = cms::cuda::make_device_unique<float[]>(NUM_VALUES * NUM_VALUES, stream);
  dim3 threadsPerBlock3{NUM_VALUES, NUM_VALUES};
  dim3 blocksPerGrid3{1, 1};
  if (NUM_VALUES * NUM_VALUES > 32) {
    threadsPerBlock3.x = 32;
    threadsPerBlock3.y = 32;
    blocksPerGrid3.x = ceil(double(NUM_VALUES) / double(threadsPerBlock3.x));
    blocksPerGrid3.y = ceil(double(NUM_VALUES) / double(threadsPerBlock3.y));
  }
  vectorProd<<<blocksPerGrid3, threadsPerBlock3, 0, stream>>>(d_a.get(), d_b.get(), d_ma.get(), NUM_VALUES);
  vectorProd<<<blocksPerGrid3, threadsPerBlock3, 0, stream>>>(d_a.get(), d_c.get(), d_mb.get(), NUM_VALUES);
  matrixMul<<<blocksPerGrid3, threadsPerBlock3, 0, stream>>>(d_ma.get(), d_mb.get(), d_mc.get(), NUM_VALUES);

  matrixMulVector<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_mc.get(), d_b.get(), d_c.get(), NUM_VALUES);

  return d_a;
}
